
#include <hip/hip_runtime.h>
#include <iostream>
#include <bits/stdc++.h>
using namespace std;
class Stack
{
private:
    int Size;
    int* arr;
public:
    Stack();
    ~Stack();
    void Append(int x);
    void Pop();
    void Destroy();
    void Peek();
    void Show();
};
Stack::Stack() : Size(1)
{
    arr = new int[INT_MAX];
}
Stack::~Stack()
{
    arr = nullptr;
}
void Stack::Append(int x)
{
    arr[Size-1] = x;
    Size++;
}
void Stack::Pop()
{
    arr[Size-1] = -1;
    Size--;
}
void Stack::Destroy()
{
    this->~Stack();
}

void Stack::Peek()
{
    cout << this->arr[Size-1] << endl;
}
void Stack::Show()
{
    cout << '[';
    for(int i=Size-2;i>0;i--)
    {
        cout << this->arr[i] << ',';
    }
    cout << this->arr[0] << ']' << endl;
}

